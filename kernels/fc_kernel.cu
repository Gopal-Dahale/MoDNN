#include "hip/hip_runtime.h"
#include "../layers/layers.h"

__global__ void matrixMultiplyShared(float * A, float * B, float * C,
                                    int numARows, int numAColumns,
                                    int numBRows, int numBColumns,
                                    int numCRows, int numCColumns)
{
    __shared__ float sA[TILE_SIZE][TILE_SIZE];   // Tile size to store elements in shared memory
    __shared__ float sB[TILE_SIZE][TILE_SIZE];

    int Row = blockDim.y*blockIdx.y + threadIdx.y; //To generate ids of threads.
    int Col = blockDim.x*blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;

    for (int k = 0; k < (((numAColumns - 1)/ TILE_SIZE) + 1); k++)
    {
        if ( (Row < numARows) && (threadIdx.x + (k*TILE_SIZE)) < numAColumns)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
        {
            sA[threadIdx.y][threadIdx.x] = A[(Row*numAColumns) + threadIdx.x + (k*TILE_SIZE)];
        }
        else
        {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }
        if ( Col < numBColumns && (threadIdx.y + k*TILE_SIZE) < numBRows)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
        {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*TILE_SIZE)*numBColumns + Col];
        }
        else
        {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        for (int j = 0; j < TILE_SIZE; ++j)//Multiplying Elements present in tile
        {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
    }
    if (Row < numCRows && Col < numCColumns)//Saving Final result into Matrix C
    {
        C[Row*numCColumns + Col] = Cvalue;
    }
}


__global__ void matrixMultiplyNaive(float * A, float * B, float * C,
                                    int N,int K,int M)
{

    int Row = blockDim.y*blockIdx.y + threadIdx.y; //To generate ids of threads.
    int Col = blockDim.x*blockIdx.x + threadIdx.x;

    if(Row<N && Col<M)
    {
      float Cvalue = 0.0;
      int k;
      for(k=0;k<K;k++)
      {
        Cvalue += A[Row*K+k] * B[k*M+Col];
      }
      C[Row*M+Col] = Cvalue;
    }
}
