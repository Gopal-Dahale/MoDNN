#include "hip/hip_runtime.h"
#include <iostream>
#include "layers.h"
#include <random>

using namespace layers;

std::map<std::string,float*> init_buffer_map()
{
  std::map<std::string,float*> buffer_map;
  buffer_map["input"] = nullptr;
  buffer_map["output"] = nullptr;
  buffer_map["workspace"] = nullptr;
  buffer_map["params"] = nullptr;

  return buffer_map;
}


ConvLayer::ConvLayer(hipdnnHandle_t cudnn,
                  int batch_size,
                  int input_height,
                  int input_width,
                  int input_channels,
                  int kernel_height,
                  int kernel_width,
                  int output_channels,
                  padding_type pad
           )
  {
    handle = cudnn;
    ibatch_size = batch_size;
    ichannels = input_channels;
    iheight = input_height;
    iwidth = input_width;
    ikernel_width = kernel_width;
    ikernel_height = kernel_height;

    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                          /*format=*/HIPDNN_TENSOR_NCHW,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/batch_size,
                                          /*channels=*/input_channels,
                                          /*image_height=*/input_height,
                                          /*image_width=*/input_width));
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));

    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*format=*/HIPDNN_TENSOR_NCHW,
                                          /*out_channels=*/output_channels,
                                          /*in_channels=*/input_channels,
                                          /*kernel_height=*/kernel_height,
                                          /*kernel_width=*/kernel_width));

    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));

    if(pad == SAME){
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                               /*pad_height=*/kernel_height/2,
                                               /*pad_width=*/kernel_width/2,
                                               /*vertical_stride=*/1,
                                               /*horizontal_stride=*/1,
                                               /*dilation_height=*/1,
                                               /*dilation_width=*/1,
                                               /*mode=*/HIPDNN_CROSS_CORRELATION,
                                               /*computeType=*/HIPDNN_DATA_FLOAT));
   }
   else if(pad == VALID)
   {
     checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                                /*pad_height=*/0,
                                                /*pad_width=*/0,
                                                /*vertical_stride=*/1,
                                                /*horizontal_stride=*/1,
                                                /*dilation_height=*/1,
                                                /*dilation_width=*/1,
                                                /*mode=*/HIPDNN_CROSS_CORRELATION,
                                                /*computeType=*/HIPDNN_DATA_FLOAT));
   }

   obatch_size= 0, ochannels= 0, oheight = 0, owidth = 0;
   checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                    input_descriptor,
                                                    kernel_descriptor,
                                                    &obatch_size,
                                                    &ochannels,
                                                    &oheight,
                                                    &owidth));

   //std::cerr << "Output Image: " << obatch_size << " x "<< oheight << " x " << owidth << " x " << ochannels
  //           << std::endl;



   checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
   checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                         /*format=*/HIPDNN_TENSOR_NCHW,
                                         /*dataType=*/HIPDNN_DATA_FLOAT,
                                         /*batch_size=*/obatch_size,
                                         /*channels=*/ochannels,
                                         /*image_height=*/oheight,
                                         /*image_width=*/owidth));

   checkCUDNN(
       hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                           input_descriptor,
                                           kernel_descriptor,
                                           convolution_descriptor,
                                           output_descriptor,
                                           HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                           /*memoryLimitInBytes=*/0,
                                           &convolution_algorithm));

   checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                      input_descriptor,
                                                      kernel_descriptor,
                                                      convolution_descriptor,
                                                      output_descriptor,
                                                      convolution_algorithm,
                                                      &forward_workspace_bytes));


    //std::cerr << "Forward Workspace Size: " << (forward_workspace_bytes / 1048576.0) << "MB"
    //          << std::endl;



    //Filter Backward Algorithm and Workspace Size

    size_t temp;
    backward_workspace_bytes=0;



    checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(
              handle, input_descriptor, output_descriptor, convolution_descriptor, kernel_descriptor,
              HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &filter_algo));


    checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
              handle, input_descriptor, output_descriptor, convolution_descriptor, kernel_descriptor,
              filter_algo, &temp));



    backward_workspace_bytes = std::max(temp,backward_workspace_bytes);


    //Data Backward Algorithm and workspace size

    checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(
              handle, kernel_descriptor, output_descriptor, convolution_descriptor, input_descriptor,
              HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &data_algo));

    checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(
        handle, kernel_descriptor, output_descriptor, convolution_descriptor, input_descriptor,
        data_algo, &temp));

    backward_workspace_bytes = std::max(temp,backward_workspace_bytes);
    //std::cerr << "Backward Workspace Size: " << (backward_workspace_bytes / 1048576.0) << "MB"
    //          << std::endl;
  }


int ConvLayer::get_output_shape_and_bytes(int shape[])
  {
    //Get Output Shape in NHWC format
    shape[0] = obatch_size;
    shape[1] = oheight;
    shape[2] = owidth;
    shape[3] = ochannels;
    return shape[0]*shape[1]*shape[2]*shape[3]*sizeof(float);
  }


void Layer::forward()
{

}

int ConvLayer::get_input_shape_and_bytes(int shape[])
  {
    //Get Output Shape in NHWC format
    shape[0] = ibatch_size;
    shape[1] = iheight;
    shape[2] = iwidth;
    shape[3] = ichannels;
    return shape[0]*shape[1]*shape[2]*shape[3]*sizeof(float);
  }

size_t ConvLayer::get_forward_workspace_bytes()
  {
    return forward_workspace_bytes;
  }

size_t ConvLayer::get_backward_workspace_bytes()
  {
    return backward_workspace_bytes;
  }

size_t ConvLayer::get_total_workspace_size()
  {
    return std::max(forward_workspace_bytes,backward_workspace_bytes);
  }

void ConvLayer::forward(float alpha, float beta, float* d_input, float* d_kernel, void* d_workspace, float * d_output)
  {
    checkCUDNN(hipdnnConvolutionForward(handle,
                                       &alpha,
                                       input_descriptor,
                                       d_input,
                                       kernel_descriptor,
                                       d_kernel,
                                       convolution_descriptor,
                                       convolution_algorithm,
                                       d_workspace,
                                       forward_workspace_bytes,
                                       &beta,
                                       output_descriptor,
                                       d_output));
  }

int ConvLayer::allocate_internal_mem(float **d_kernel, void **d_workspace)
  {
      int param_size = sizeof(float)*ikernel_width*ikernel_height*ichannels*ochannels;
      int workspace_size = get_total_workspace_size();
      hipMalloc(d_kernel, param_size);
      hipMalloc(d_workspace,workspace_size);

      return param_size+workspace_size;

  }

void ConvLayer::populate_filter_params(float *d_kernel)
{
  float init_params[ochannels][ikernel_height][ikernel_width][ichannels];
  std::normal_distribution<float> distribution(0,0.01);
  std::default_random_engine generator;

  for(int ochannel = 0; ochannel < ochannels; ochannel++)
    for(int row=0;row<ikernel_height;row++)
      for(int col=0;col<ikernel_width;col++)
        for(int ichannel=0;ichannel < ichannels; ichannel++)
          init_params[ochannel][row][col][ichannel] = distribution(generator);


  hipMemcpy(d_kernel,init_params,sizeof(init_params),hipMemcpyHostToDevice);

}



ConvLayer::~ConvLayer()
{
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
}


InputLayer::InputLayer(int batch_size, int height, int width, int channels,int num_classes)
{
  ibatch_size = obatch_size = batch_size;
  iheight = oheight = height;
  iwidth = owidth = width;
  ichannels = ochannels = channels;
  this->num_classes = num_classes;
}

int InputLayer::get_output_shape_and_bytes(int shape[])
{
    //Get Output Shape in NHWC format
    shape[0] = obatch_size;
    shape[1] = oheight;
    shape[2] = owidth;
    shape[3] = ochannels;
    return shape[0]*shape[1]*shape[2]*shape[3]*sizeof(float);
}



void InputLayer::randomly_populate(float *data,float * labels)
{
  float init_params[obatch_size][oheight][owidth][ochannels],init_labels[obatch_size][num_classes];
  int class_;
  std::normal_distribution<float> distribution(0,0.01);
  std::default_random_engine generator;

  for(int data_point = 0; data_point < obatch_size; data_point++)
    for(int row=0;row<oheight;row++)
      for(int col=0;col<owidth;col++)
        for(int ochannel=0;ochannel < ochannels; ochannel++){
          init_params[data_point][row][col][ochannel] = distribution(generator);
        }

  //std::cout << "Checking random input layer" << std::endl;
  //std::cout << init_params[0][0][0][1] << std::endl;
  hipMemcpy(data,init_params,sizeof(init_params),hipMemcpyHostToDevice);

  for (int j = 0; j < obatch_size; j++)
  {
    class_ = rand() % num_classes;

    for (int i = 0; i < num_classes; i++)
      init_labels[j][i] = class_ == i ? -num_classes / (float) obatch_size : 0;
  }

  hipMemcpy(labels,init_labels,sizeof(init_labels),hipMemcpyHostToDevice);

}


Flatten::Flatten(int batch_size,int input_height,int input_width,int input_channels)
{
  ibatch_size = batch_size;
  ichannels = input_channels;
  iheight = input_height;
  iwidth = input_width;
  obatch_size = batch_size;
  oheight = input_channels*input_height*input_width;

}

int Flatten::get_output_shape_and_bytes(int shape[])
{
  shape[0] = obatch_size;
  shape[1] = oheight;
  shape[2] = -1;
  shape[3] = -1;



  return obatch_size*oheight*sizeof(float);
}


FCLayer::FCLayer(hipblasHandle_t cublas,int batch_size, int input_height, int output_height)
{
  handle = cublas;
  ibatch_size = batch_size;
  iheight = input_height;
  obatch_size = ibatch_size;
  oheight = output_height;
}

int FCLayer::get_output_shape_and_bytes(int shape[])
{
  shape[0] = obatch_size;
  shape[1] = oheight;
  shape[2] = -1;
  shape[3] = -1;

  return obatch_size*oheight*sizeof(float);
}

int FCLayer::allocate_internal_mem(float **d_kernel)
{
  int param_size = iheight*oheight*sizeof(float);
  hipMalloc(d_kernel,param_size);
  return param_size;
}

void FCLayer::populate_filter_params(float *d_kernel)
{
  float init_params[iheight][oheight];
  std::normal_distribution<float> distribution(0,0.01);
  std::default_random_engine generator;
  for(int i=0;i<iheight;i++)
    for(int j=0;j<oheight;j++)
      init_params[i][j] = distribution(generator);



  hipMemcpy(d_kernel,init_params,sizeof(init_params),hipMemcpyHostToDevice);
}

void FCLayer::forward(float * d_input, float * d_kernel, float * d_output)
{
  float alpha = 1.0,beta = 0.0;
  hipblasSgemm(handle,
              HIPBLAS_OP_N,
              HIPBLAS_OP_N,
              oheight,/*m*/
              obatch_size,/*n*/
              iheight,/*k*/
              &alpha,
              d_kernel,
              oheight,
              d_input,
              iheight,
              &beta,
              d_output,
              oheight
            );
}

Softmax::Softmax(hipdnnHandle_t cudnn,int batch_size,int input_height)
{
  handle = cudnn;
  ibatch_size = obatch_size = batch_size;
  iheight = oheight = input_height;
  hipdnnCreateTensorDescriptor(&input_descriptor);
  hipdnnCreateTensorDescriptor(&output_descriptor);
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,HIPDNN_TENSOR_NCHW,HIPDNN_DATA_FLOAT,ibatch_size,iheight,1,1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,HIPDNN_TENSOR_NCHW,HIPDNN_DATA_FLOAT,ibatch_size,iheight,1,1));
}

int Softmax::get_output_shape_and_bytes(int shape[])
{
  shape[0] = obatch_size;
  shape[1] = oheight;
  shape[2] = -1;
  shape[3] = -1;

  return obatch_size*oheight*sizeof(float);
}

void Softmax::forward(float* d_input, float * d_output)
{
  float alpha = 1.0;
  float beta = 0.0;
  hipdnnSoftmaxForward(handle,
                      HIPDNN_SOFTMAX_ACCURATE,
                      HIPDNN_SOFTMAX_MODE_CHANNEL,
                      &alpha,
                      input_descriptor,
                      d_input,
                      &beta,
                      output_descriptor,
                      d_output);
}
