#include "fc_layer.h"

using namespace layers;

FCLayer::FCLayer(hipblasHandle_t cublas,int batch_size, int input_height, int output_height)
{
  handle = cublas;
  ibatch_size = batch_size;
  iheight = input_height;
  obatch_size = ibatch_size;
  oheight = output_height;
}

int FCLayer::get_output_shape_and_bytes(int shape[])
{
  shape[0] = obatch_size;
  shape[1] = oheight;
  shape[2] = -1;
  shape[3] = -1;

  return obatch_size*oheight*sizeof(float);
}

int FCLayer::get_input_shape_and_bytes(int shape[])
{
  shape[0] = obatch_size;
  shape[1] = iheight;
  shape[2] = -1;
  shape[3] = -1;

  return obatch_size*oheight*sizeof(float);
}

int FCLayer::get_params_shape_and_bytes(int shape[])
{
  shape[0] = iheight;
  shape[1] = oheight;
  shape[2] = -1;
  shape[3] = -1;

  return iheight*oheight*sizeof(float);
}

int FCLayer::allocate_internal_mem(float **d_kernel,float **d_diffkernel)
{
  int param_size = iheight*oheight*sizeof(float);
  hipMalloc(d_kernel,param_size);
  hipMalloc(d_diffkernel,param_size);
  return param_size;
}

void FCLayer::populate_filter_params(float *d_kernel)
{
  float* init_params = (float*) malloc(iheight*oheight*sizeof(float));
  std::normal_distribution<float> distribution(MU,SIGMA);
  std::default_random_engine generator;
  for(int i=0;i<iheight;i++)
    for(int j=0;j<oheight;j++)
      init_params[i*oheight+j] = distribution(generator);

  hipMemcpy(d_kernel,init_params,iheight*oheight*sizeof(float),hipMemcpyHostToDevice);
}

void FCLayer::forward(float * d_input, float * d_kernel, float * d_output)
{
  float alpha = 1.0,beta = 0.0;
  //we are multiplying d_input(A) * d_kernel(B), both stored in row major form
  //d_input [obatch_sizexiheight] d_okernel[iheight*oheight]
  //in comments A[MxK] B[KxN]
  checkCUBLAS(hipblasSgemm(handle,
              HIPBLAS_OP_N, //info for B, use HIPBLAS_OP_T if you want to use BT
              HIPBLAS_OP_N, //info for A, use HIPBLAS_OP_T if you want to use AT
              oheight,/*N*/
              obatch_size,/*M*/
              iheight,/*K*/
              &alpha,
              d_kernel, //B
              oheight, //N
              d_input, //A
              iheight, //K
              &beta,
              d_output,//C
              oheight //K
            ));
}

void FCLayer::backward(float *d_input, float* d_kernel,float *d_diffkernel,float *d_diffinput, float *d_diffoutput)
{
  float alpha = 1.0,beta = 0.0;
  checkCUBLAS(hipblasSgemm(handle,
              HIPBLAS_OP_N, //info for B, use HIPBLAS_OP_T if you want to use BT
              HIPBLAS_OP_T, //info for A, use HIPBLAS_OP_T if you want to use AT
              oheight,/*N*/
              iheight,/*M*/
              obatch_size,/*K*/
              &alpha,
              d_diffoutput, //B
              oheight, //N
              d_input, //A
              iheight, //K
              &beta,
              d_diffkernel,//C
              oheight //K
            ));

  checkCUBLAS(hipblasSgemm(handle,
              HIPBLAS_OP_T, //info for B, use HIPBLAS_OP_T if you want to use BT
              HIPBLAS_OP_N, //info for A, use HIPBLAS_OP_T if you want to use AT
              iheight,/*N*/
              obatch_size,/*M*/
              oheight,/*K*/
              &alpha,
              d_kernel, //B
              oheight, //N
              d_diffoutput, //A
              oheight, //K
              &beta,
              d_diffinput,//C
              iheight //K
            ));
}


