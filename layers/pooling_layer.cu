#include "pooling_layer.h"

using namespace layers;

PoolingLayer::PoolingLayer(hipdnnHandle_t* handle,
    int window_height,
    int window_width,
    int vertical_stride,
    int horizontal_stride,
    int batch_size,
    int input_height,
    int input_width,
    int input_channels,
    padding_type pad,
    hipdnnPoolingMode_t mode) {
    // mode -- HIPDNN_POOLING_MAX(0), HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING(1)
    //                 HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING(2), HIPDNN_POOLING_MAX_DETERMINISTIC(3)

    handle_ = handle;
    ibatch_size = batch_size;
    ichannels = input_channels;
    iheight = input_height;
    iwidth = input_width;

    checkCUDNN(hipdnnCreatePoolingDescriptor(&pooling_descriptor));
    if (pad == SAME) {
        checkCUDNN(hipdnnSetPooling2dDescriptor(
            /*poolingDesc=*/pooling_descriptor,
            /*mode=*/mode,
            /*maxpoolingNanOpt=*/HIPDNN_NOT_PROPAGATE_NAN,
            /*windowHeight*/window_height,
            /*windowWidth*/window_width,
            /*verticalPadding*/window_height/2,
            /*horizontalPadding*/window_width/2,
            /*verticalStride*/vertical_stride,
            /*horizontalStride*/horizontal_stride
        ));
    } else if (pad == VALID) {
        checkCUDNN(hipdnnSetPooling2dDescriptor(
            /*poolingDesc=*/pooling_descriptor,
            /*mode=*/mode,
            /*maxpoolingNanOpt=*/HIPDNN_NOT_PROPAGATE_NAN,
            /*windowHeight*/window_height,
            /*windowWidth*/window_width,
            /*verticalPadding*/0,
            /*horizontalPadding*/0,
            /*verticalStride*/vertical_stride,
            /*horizontalStride*/horizontal_stride
        ));
    }

    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
        /*format=*/HIPDNN_TENSOR_NCHW,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/ibatch_size,
        /*channels=*/ichannels,
        /*image_height=*/iheight,
        /*image_width=*/iwidth));

    checkCUDNN(hipdnnGetPooling2dForwardOutputDim(pooling_descriptor,
        input_descriptor,
        &obatch_size,
        &ochannels,
        &oheight,
        &owidth));

    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
        /*format=*/HIPDNN_TENSOR_NCHW,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/obatch_size,
        /*channels=*/ochannels,
        /*image_height=*/oheight,
        /*image_width=*/owidth
    ));
}

void PoolingLayer::forward(float alpha, float beta, float* d_input, float* d_output) {
    checkCUDNN(hipdnnPoolingForward(*handle_,
        pooling_descriptor,
        &alpha,
        input_descriptor,
        d_input,
        &beta,
        output_descriptor,
        d_output
    ));
}

void PoolingLayer::backward(float alpha, float beta, float* d_y, float* d_dy, float* d_x, float* d_dx) {
    checkCUDNN(hipdnnPoolingBackward(*handle_,
        pooling_descriptor,
        &alpha,
        output_descriptor,
        d_y,
        output_descriptor,
        d_dy,
        input_descriptor,
        d_x,
        &beta,
        input_descriptor,
        d_dx
    ));
}

int PoolingLayer::get_output_shape_and_bytes(int shape[]) {
    shape[0] = obatch_size;
    shape[1] = oheight;
    shape[2] = owidth;
    shape[3] = ochannels;
    return sizeof(float)*obatch_size*ochannels*oheight*owidth;
}

int PoolingLayer::get_total_memory()
{
  int shape[4];
  return get_output_shape_and_bytes(shape);
}

PoolingLayer::~PoolingLayer() {
    checkCUDNN(hipdnnDestroyTensorDescriptor(input_descriptor));
	checkCUDNN(hipdnnDestroyTensorDescriptor(output_descriptor));
    checkCUDNN(hipdnnDestroyPoolingDescriptor(pooling_descriptor));
}
