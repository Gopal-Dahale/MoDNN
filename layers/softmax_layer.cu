#include "softmax_layer.h"

using namespace layers;

Softmax::Softmax(hipdnnHandle_t cudnn,int batch_size,int input_height)
{
  handle = cudnn;
  ibatch_size = obatch_size = batch_size;
  iheight = oheight = input_height;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&diff_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptorEx(input_descriptor,HIPDNN_DATA_FLOAT,obatch_size,oheight,1,1,oheight,1,1,1));
  checkCUDNN(hipdnnSetTensor4dDescriptorEx(output_descriptor,HIPDNN_DATA_FLOAT,obatch_size,oheight,1,1,oheight,1,1,1));
  checkCUDNN(hipdnnSetTensor4dDescriptorEx(diff_descriptor,HIPDNN_DATA_FLOAT,obatch_size,oheight,1,1,oheight,1,1,1));

}

int Softmax::get_output_shape_and_bytes(int shape[])
{
  shape[0] = obatch_size;
  shape[1] = oheight;
  shape[2] = -1;
  shape[3] = -1;

  return obatch_size*oheight*sizeof(float);
}

int Softmax::get_input_shape_and_bytes(int shape[])
{
  shape[0] = obatch_size;
  shape[1] = oheight;
  shape[2] = -1;
  shape[3] = -1;

  return obatch_size*oheight*sizeof(float);
}

void Softmax::forward(float* d_input, float * d_output)
{
  float alpha = 1.0;
  float beta = 0.0;
  checkCUDNN(hipdnnSoftmaxForward(handle,
                      HIPDNN_SOFTMAX_ACCURATE,
                      HIPDNN_SOFTMAX_MODE_CHANNEL,
                      &alpha,
                      input_descriptor,
                      d_input,
                      &beta,
                      output_descriptor,
                      d_output));

}


void Softmax::backward(const int *label, float *diff, float * output)
{
  hipMemcpy(diff,output,obatch_size*oheight*sizeof(float),hipMemcpyDeviceToDevice);
  SoftmaxLossBackprop<<<(obatch_size+255)/256, 256>>>(label, oheight, obatch_size, diff);
}
