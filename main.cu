#include "hip/hip_runtime.h"
#include "layers/layers.h"
#include <fstream>
#include <math.h>
#include "mnist_dataset/mnist.h"
#include "data_core/data_loader.h"
#include "trainer/trainer.h"
#include <hip/hip_runtime_api.h>

using namespace layers;
using namespace network;




int main(int argc, const char* argv[])
{
    // main function for executing the code
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    hipblasHandle_t cublas;
    hipblasCreate(&cublas);
    std::ofstream outdata;





    // hipSetDevice(0);

    std::string images_file_str = "mnist_dataset/data/train-images.idx3-ubyte";
    std::string label_file_str = "mnist_dataset/data/train-labels.idx1-ubyte";
    char * images_file = (char*)images_file_str.c_str();
    char * label_file = (char*)label_file_str.c_str();
    std::cout << images_file << " "<<label_file << std::endl;
    float* data_batch, *label_batch;
    unsigned batch_size = BATCH_SIZE,rows, sub_batch_size;
    unsigned dataset_size, offset;

    std::cout << "Creating Dataset" << std::endl;
    Dataset* dataset= new MNIST(images_file, label_file, true);
    dataset_size = dataset->getDatasetSize();

    std::cout << "Creating DataLoader" << std::endl;

    DataLoader* dataloader = new DataLoader(dataset, batch_size);
    rows = sqrt(dataset->getInputDim());
    std::string input_spec = "input "  + std::to_string(batch_size)+ " " + std::to_string(rows) +" "+std::to_string(rows)+ " " + "1 " +std::to_string(dataset->getLabelDim());
    int* label_batch_integer = (int*)malloc(sizeof(int)*batch_size);



    //std::vector<std::string> specs = {input_spec,"flatten","fc "+std::to_string(dataset->getLabelDim()),"softmax"};
    //le net specification
    std::vector<std::string> specs = {input_spec,
                                      "conv 5 5 20",
                                      "relu",
                                      "maxpool 2 2 2 2",
                                      "conv 5 5 50",
                                      "relu",
                                      "maxpool 2 2 2 2",
                                      "flatten",
                                      "fc 500",
                                      "relu",
                                      "fc "+std::to_string(dataset->getLabelDim()),
                                      "softmax"};



    // int MAX_MEM = 105458240; //25MB
    // int USE_MEM = MAX_MEM; //8MB
    seqNetwork * nn = new seqNetwork(cudnn,cublas,specs,LR,0,0);
    std::cout << (float)nn->get_total_memory()/1000000 << " MB " <<std::endl;
    vmm * mem_manager = new vmm(2*nn->get_total_memory(),&(nn->layer_buffers));


    // hipProfilerStart();
     train_with_minimal_memory(dataloader,dataset,nn, mem_manager,5);
    // train_with_full_memory(dataloader,dataset,nn,mem_manager,5);
    // hipProfilerStop();
    return 0;

}
