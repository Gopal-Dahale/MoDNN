#include "hip/hip_runtime.h"
#include "layers/layers.h"
#include <fstream>
#include <math.h>
#include "mnist_dataset/mnist.h"
#include "data_core/data_loader.h"

using namespace layers;
using namespace network;



cv::Mat load_image(const char* image_path) {
  cv::Mat image = cv::imread(image_path, cv::IMREAD_COLOR );
  image.convertTo(image, CV_32FC3);
  cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
  std::cerr << "Input Image: " << image.rows << " x " << image.cols << " x "
            << image.channels() << std::endl;
  return image;
}

void save_image(const char* output_filename,
                float* buffer,
                int height,
                int width,
                int channels) {
  cv::Mat output_image(height, width, CV_32FC3, buffer);
  // Make negative values zero.
  cv::threshold(output_image,
                output_image,
                /*threshold=*/0,
                /*maxval=*/0,
                cv::THRESH_TOZERO);
  cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
  output_image.convertTo(output_image, CV_8UC3);
  cv::imwrite(output_filename, output_image);
  std::cerr << "Wrote output to " << output_filename << std::endl;
}

void print_output(float * layer,int shape[])
{
  for(int i=0;i<shape[0];i++){
    for(int j=0;j<shape[1]*shape[2]*shape[3];j++){
        std::cout << layer[i*shape[1]*shape[2]*shape[3]+j] << " ";
    }
    std::cout << std::endl;
  }

}

float categorical_cross_entropy_loss(float * softmax_dinput,int shape[])
{
  float temp,loss=0;
  for(int i=0;i<shape[0];i++){
    for(int j=0;j<shape[1];j++){
        temp = softmax_dinput[i*shape[1]+j];
        if(temp<=0)
        {
          temp = temp+1;
          loss += -log(temp);
          break;
        }
    }

  }
  return loss/shape[0];
}


void label_batch_converter_mnist(float* batch, int* batch_target, unsigned batch_size) {
  for (int i = 0; i < batch_size; i++) {
    batch_target[i] = batch[i];
  }
}

int main(int argc, const char* argv[])
{
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    hipblasHandle_t cublas;
    hipblasCreate(&cublas);
    std::ofstream outdata;

    char* images_file = "mnist_dataset/data/train-images.idx3-ubyte";
    char* label_file = "mnist_dataset/data/train-labels.idx1-ubyte";
    float* data_batch, *label_batch;
    unsigned batch_size = 2,rows;
    Dataset* dataset= new MNIST(images_file, label_file, true);
    DataLoader* dataloader = new DataLoader(dataset, batch_size);

    rows = sqrt(dataset->getInputDim());
    std::string input_spec = "input "  + std::to_string(batch_size)+ " " + std::to_string(rows) +" "+std::to_string(rows)+ " " + "1 " +std::to_string(dataset->getLabelDim());
    std::cout << input_spec << std::endl;
    std::vector<std::string> specs = {input_spec,"conv 3 3 3","relu","maxpool 2 2 2 2","flatten","fc 50","relu","fc "+std::to_string(dataset->getLabelDim()),"softmax"};
    
    int* label_batch_integer = (int*)malloc(sizeof(int)*batch_size);
    seqNetwork nn = seqNetwork(cudnn,cublas,specs,LR);
    nn.print_network_info();
    nn.allocate_memory();
    int shape[4];
    nn.get_output_shape(shape,nn.num_layers-1);

    std::cout << "Printing output shape of Neural Network" << std::endl;
    for(int i=0;i<4;i++)
      std::cout << shape[i] <<" "<<" ";
    std::cout<<std::endl;

    // std::cout << "Randomising input to the neural network" << std::endl;
    // nn.randomise_batch();

    std::cout << "Randomising Parameters of the neural network" << std::endl;
    nn.randomise_params();

    std::cout << "Forward Pass for the neural network" << std::endl;

    float * output,loss;
    for(int i=0;i<10000;i++)
    {
      dataloader->get_next_batch(&data_batch, &label_batch);
      label_batch_converter_mnist(label_batch, label_batch_integer, batch_size);
      nn.update_batch(data_batch, label_batch_integer);
      nn.forward();
      nn.backward();
      if(i%1000==0){
      output = nn.offload_buffer(nn.num_layers-1,"dinput",shape);
      loss = categorical_cross_entropy_loss(output,shape);
      std::cout << "Iteration number "<<i<<" CCE Loss :- "<<loss <<std::endl;
      }
    }


    //test for relu - passed
    // nn.forward();
    // output = nn.offload_buffer(3,"output",shape);
    // print_output(output,shape);
    // nn.backward();
    // output = nn.offload_buffer(3,"doutput",shape);
    // print_output(output,shape);
    // output = nn.offload_buffer(3,"dinput",shape);
    // print_output(output,shape);
    return 0;

}
