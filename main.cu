#include "hip/hip_runtime.h"
#include "layers/layers.h"
#include <fstream>
#include <math.h>
#include "mnist_dataset/mnist.h"
#include "data_core/data_loader.h"

using namespace layers;
using namespace network;



// cv::Mat load_image(const char* image_path) {
//   cv::Mat image = cv::imread(image_path, cv::IMREAD_COLOR );
//   image.convertTo(image, CV_32FC3);
//   cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
//   std::cerr << "Input Image: " << image.rows << " x " << image.cols << " x "
//             << image.channels() << std::endl;
//   return image;
// }
//
// void save_image(const char* output_filename,
//                 float* buffer,
//                 int height,
//                 int width,
//                 int channels) {
//   cv::Mat output_image(height, width, CV_32FC3, buffer);
//   // Make negative values zero.
//   cv::threshold(output_image,
//                 output_image,
//                 /*threshold=*/0,
//                 /*maxval=*/0,
//                 cv::THRESH_TOZERO);
//   cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
//   output_image.convertTo(output_image, CV_8UC3);
//   cv::imwrite(output_filename, output_image);
//   std::cerr << "Wrote output to " << output_filename << std::endl;
// }

void print_output(float * layer,int shape[])
{
  for(int i=0;i<shape[0];i++){
    for(int j=0;j<shape[1]*shape[2]*shape[3];j++){
        std::cout << layer[i*shape[1]*shape[2]*shape[3]+j] << " ";
    }
    std::cout << std::endl;
  }

}

float categorical_cross_entropy_loss(float * softmax_dinput,int shape[], int * label_batch_integer)
{
  float temp,loss=0;
  for(int i=0;i<shape[0];i++){
      int j = label_batch_integer[i];
      temp = softmax_dinput[i*shape[1]+j];
      temp = temp+1;
      loss += -log(temp);

  }
  return loss;
}


void label_batch_converter_mnist(float* batch, int* batch_target, unsigned batch_size)
{
  for (int i = 0; i < batch_size; i++)
  {
    batch_target[i] = int(batch[i]);
  }
}

int main(int argc, const char* argv[])
{
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    hipblasHandle_t cublas;
    hipblasCreate(&cublas);
    std::ofstream outdata;


    std::string images_file_str = "/content/src/mnist_dataset/data/train-images-idx3-ubyte";
    std::string label_file_str = "/content/src/mnist_dataset/data/train-labels-idx1-ubyte";
    char * images_file = (char*)images_file_str.c_str();
    char * label_file = (char*)label_file_str.c_str();
    std::cout << images_file << " "<<label_file << std::endl;
    float* data_batch, *label_batch;
    unsigned batch_size = 20,rows;
    Dataset* dataset= new MNIST(images_file, label_file, true);
    DataLoader* dataloader = new DataLoader(dataset, batch_size);
    rows = sqrt(dataset->getInputDim());
    std::string input_spec = "input "  + std::to_string(batch_size)+ " " + std::to_string(rows) +" "+std::to_string(rows)+ " " + "1 " +std::to_string(dataset->getLabelDim());
    int* label_batch_integer = (int*)malloc(sizeof(int)*batch_size);


    std::vector<std::string> specs = {input_spec,"conv 3 3 3","relu","maxpool 2 2 2 2","flatten","fc 50","relu","fc "+std::to_string(dataset->getLabelDim()),"softmax"};
    seqNetwork nn = seqNetwork(cudnn,cublas,specs,LR);

    vmm * mem_manager = new vmm(nn.get_total_memory()+20);
    nn.allocate_all_memory(mem_manager);

    mem_manager->printNodes();

    nn.print_network_info();
    int shape[4];
    nn.get_output_shape(shape,nn.num_layers-1);

    std::cout << "Printing output shape of Neural Network" << std::endl;
    for(int i=0;i<4;i++)
      std::cout << shape[i] <<" "<<" ";
    std::cout<<std::endl;


    std::cout << "Randomising Parameters of the neural network" << std::endl;
    nn.randomise_params();

    std::cout << "Forward Pass for the neural network" << std::endl;

    float * output,loss;

    int epochs=50;
    int num_iters_in_epoch =  dataset->getDatasetSize()/batch_size;
    bool rem = false;


    if(dataset->getDatasetSize()%batch_size!=0){
      num_iters_in_epoch+=1;
      rem = true;
      }
    if(rem)
      std::cout << "Ignoring last batch " << std::endl;

    std::cout << "Number of iterations in an epoch " << num_iters_in_epoch << std::endl;
    for(int j=0;j<epochs;j++)
    {
      loss = 0;
      for(int i=0;i<num_iters_in_epoch;i++)
      {
        if(rem && i==num_iters_in_epoch-1)
          break;

        dataloader->get_next_batch(&data_batch, &label_batch);
        label_batch_converter_mnist(label_batch, label_batch_integer, batch_size);


        nn.update_batch(data_batch, label_batch_integer);
        nn.forward();
        nn.backward();

        if(j%10==0)
        {
          output = nn.offload_buffer(nn.num_layers-1,"dinput",shape);
          loss += categorical_cross_entropy_loss(output,shape,label_batch_integer);
        }

      }

      if(j%10==0)
      {
        loss = loss/(float)(dataset->getDatasetSize());
        std::cout << "Epoch number "<<j+1<< " : " << "Loss :- " << loss <<std::endl;
      }
      dataloader->reset();
      dataset->shuffle();
    }

    return 0;

}
