#include "hip/hip_runtime.h"
#include "layers/layers.h"
#include <fstream>
#include <math.h>
#include "mnist_dataset/mnist.h"
#include "data_core/data_loader.h"
#include "trainer/trainer.h"
#include <hip/hip_runtime_api.h>
#include <ctime>

using namespace layers;
using namespace network;




int main(int argc, const char* argv[])
{
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    hipblasHandle_t cublas;
    hipblasCreate(&cublas);
    std::ofstream outdata;





    // hipSetDevice(0);

    std::string images_file_str = "/content/src/mnist_dataset/data/train-images-idx3-ubyte";
    std::string label_file_str = "/content/src/mnist_dataset/data/train-labels-idx1-ubyte";
    char * images_file = (char*)images_file_str.c_str();
    char * label_file = (char*)label_file_str.c_str();
    std::cout << images_file << " "<<label_file << std::endl;
    float* data_batch, *label_batch;
    unsigned batch_size = BATCH_SIZE,rows, sub_batch_size;
    unsigned dataset_size, offset;

    std::cout << "Creating Dataset" << std::endl;
    Dataset* dataset= new MNIST(images_file, label_file, true);
    dataset_size = dataset->getDatasetSize();

    std::cout << "Creating DataLoader" << std::endl;

    DataLoader* dataloader = new DataLoader(dataset, batch_size);
    rows = sqrt(dataset->getInputDim());
    std::string input_spec = "input "  + std::to_string(batch_size)+ " " + std::to_string(rows) +" "+std::to_string(rows)+ " " + "1 " +std::to_string(dataset->getLabelDim());
    int* label_batch_integer = (int*)malloc(sizeof(int)*batch_size);



    //std::vector<std::string> specs = {input_spec,"flatten","fc "+std::to_string(dataset->getLabelDim()),"softmax"};
    //le net specification
    std::vector<std::string> specs = {input_spec,
                                      "conv 5 5 20",
                                      "relu",
                                      "maxpool 2 2 2 2",
                                      "conv 5 5 50",
                                      "relu",
                                      "maxpool 2 2 2 2",
                                      "flatten",
                                      "fc 500",
                                      "relu",
                                      "fc "+std::to_string(dataset->getLabelDim()),
                                      "softmax"};



    //int MAX_MEM = 0.44*62091168;
    seqNetwork * nn = new seqNetwork(cudnn,cublas,specs,LR,0,0);
    //std::cout << "Sub batch size - " << nn->sub_batch_size() << std::endl;



    std::cout << (float)nn->get_total_memory()/1000000 << " MB " <<std::endl;
    vmm * mem_manager = new vmm(nn->get_total_memory()*0.55,&(nn->layer_buffers));
    //
    time_t start = time(NULL);
    // //hipProfilerStart();
    train_with_minimal_memory(dataloader,dataset,nn, mem_manager,1);
    // // train_with_full_memory(dataloader,dataset,nn,mem_manager,5);
    // //hipProfilerStop();
    time_t end = time(NULL);
    std::cout << "Total time - "<< end-start <<"seconds"<<std::endl;
    return 0;

}
