#include "layers/pooling_layer.h"
#include "layers/input_layer.h"
#include "layers/conv_layer.h"
#include "layers/softmax_layer.h"
#include "layers/fc_layer.h"
#include "layers/flatten_layer.h"
#include "layers/layers.h"
#include "layers/relu_layer.h"


using namespace network;
using namespace layers;

seqNetwork::seqNetwork(hipdnnHandle_t cudnn,hipblasHandle_t cublas,std::vector<std::string> &specs,float lr)
{
  /*
  Specs is a vector of strings specifying the Neural Network.
  Input -> "input N H W C"
  Conv ->  "conv H W C"
  */

  num_layers = specs.size();
  handle = cudnn;
  blas_handle = cublas;
  this->lr = lr;
  for(int i=0;i<num_layers;i++)
    {
      std::stringstream ss(specs[i]);
      std::vector<std::string> info;
      std::string tok;
      while(getline(ss, tok, ' ')){
        info.push_back(tok);
      }
      layer_info.push_back(info);
    }

  make_nn_objs();

}

void seqNetwork::print_network_info()
{
  for(int i=0;i<num_layers;i++)
  {
    std::cout << "Layer "<<i+1<<" : "<<layer_info[i][0] << std::endl;
    std::cout << "Specs : ";
    for(int j=1;j<layer_info[i].size();j++)
      std::cout << layer_info[i][j] << " ";
    std::cout << std::endl;
  }
}

void seqNetwork::get_output_shape(int shape[],int i)
{
  Layer *last_layer = layer_objects[i];
  if (layer_info[i][0] == "flatten")
    ((Flatten*)last_layer)->get_output_shape_and_bytes(shape);
  else if(layer_info[i][0] == "conv")
    ((ConvLayer*)last_layer) -> get_output_shape_and_bytes(shape);
  else if(layer_info[i][0] == "fc")
    ((FCLayer*)last_layer) -> get_output_shape_and_bytes(shape);
  else if(layer_info[i][0] == "softmax")
    ((Softmax*)last_layer) -> get_output_shape_and_bytes(shape);
  else if(layer_info[i][0] == "input")
    ((InputLayer*)last_layer) -> get_output_shape_and_bytes(shape);
  else if(layer_info[i][0] == "maxpool" || layer_info[i][0] == "avgpool" )
    ((PoolingLayer*)last_layer)->get_output_shape_and_bytes(shape);
}

int seqNetwork::get_total_memory()
{
  std::map<std::string,int>::iterator it;
  std::string buff_type;
  int buff_bytes;
  int total_bytes=0;
  for(int i=0;i<num_layers;i++)
  {
    std::cout << layer_info[i][0] << std::endl;
    it = layer_buffer_bytes[i].begin();
    while(it!=layer_buffer_bytes[i].end())
    {
      buff_type = it->first;
      buff_bytes = it->second;
      total_bytes += buff_bytes;
      std::cout << buff_type << " " << buff_bytes << std::endl;
      it++;
    }
  }
  std::cout << total_bytes << std::endl;
  return total_bytes;

}

void seqNetwork::make_nn_objs()
{
  std::string layer_type;
  int shape[4],batch_size,rows,columns,channels,num_classes;
  int kernel_rows,kernel_cols,kernel_channels,bytes;
  int input_height,output_height;
  int window_height, window_width;
  int vertical_stride, horizontal_stride;
  padding_type pad;
  hipdnnPoolingMode_t pooling_type;

  //std::cout << "Allocating memory for the Neural Network" << std::endl;
  layer_buffers.resize(num_layers);
  layer_offloaded_buffers.resize(num_layers);
  layer_buffer_bytes.resize(num_layers);

  for(int i=0;i<num_layers;i++)
  {
    layer_type = layer_info[i][0];
    std::cout << "Layer "<<i+1<<" : "<<layer_type << std::endl;
    layer_offloaded_buffers[i] = init_buffer_map();
    layer_buffers[i] = init_buffer_map();
    if(layer_type == "input")
    {
      batch_size = atoi(layer_info[i][1].c_str());
      rows = atoi(layer_info[i][2].c_str());
      columns = atoi(layer_info[i][3].c_str());
      channels = atoi(layer_info[i][4].c_str());
      num_classes = atoi(layer_info[i][5].c_str());
      this->batch_size = batch_size;

      std::cout << "Setting up input layer - "<< batch_size <<" " << rows << " "<<columns <<" "<<channels << std::endl;

      InputLayer * new_ip = new InputLayer(batch_size,rows,columns,channels,num_classes);
      layer_objects.push_back(new_ip);

      bytes = new_ip->get_output_shape_and_bytes(shape);
      //layer_buffers[i] = init_buffer_map();


      layer_buffer_bytes[i]["output"]=bytes;//hipMalloc(&(layer_buffers[i]["output"]),bytes);
      layer_buffer_bytes[i]["doutput"]=bytes;//hipMalloc(&(layer_buffers[i]["doutput"]),bytes);


      layer_buffers[i]["labels"] = nullptr;
      layer_buffer_bytes[i]["labels"]=batch_size*sizeof(int);//hipMalloc(&(layer_buffers[i]["labels"]),batch_size*sizeof(int));



      //std::cout << "finished with input layer" << std::endl;
    }
    else if(layer_type == "conv")
    {
      //batch_size is already fixed in the first input layer
      //std::cout << "Allocating Memory to Conv Layer" << std::endl;
      kernel_rows = atoi(layer_info[i][1].c_str());
      kernel_cols = atoi(layer_info[i][2].c_str());
      kernel_channels = atoi(layer_info[i][3].c_str());

      this->get_output_shape(shape,i-1);

      batch_size = shape[0];
      rows = shape[1];
      columns = shape[2];
      channels = shape[3];

      std::cout << "Setting up conv layer - "<< batch_size <<" " << rows << " "<<columns <<" "<<channels << std::endl;

      ConvLayer * new_conv = new ConvLayer(handle,batch_size,rows,columns,channels,kernel_rows,kernel_cols,kernel_channels,VALID);

      bytes =  new_conv->get_output_shape_and_bytes(shape);

      layer_objects.push_back(new_conv);

      //layer_buffers[i] = init_buffer_map();

      layer_buffer_bytes[i]["output"]=bytes;//hipMalloc(&(layer_buffers[i]["output"]),bytes);
      layer_buffer_bytes[i]["doutput"]=bytes;//hipMalloc(&(layer_buffers[i]["doutput"]),bytes);

      // layer_buffers[i]["input"] = layer_buffers[i-1]["output"];
      // layer_buffers[i]["dinput"] = layer_buffers[i-1]["doutput"];


      //new_conv -> allocate_internal_mem(&(layer_buffers[i]["params"]),(void**)&(layer_buffers[i]["workspace"]),&(layer_buffers[i]["dparams"]));
      layer_buffer_bytes[i]["params"] = new_conv -> get_params_shape_and_bytes(shape);
      layer_buffer_bytes[i]["dparams"] = layer_buffer_bytes[i]["params"];
      layer_buffer_bytes[i]["workspace"] = new_conv -> get_total_workspace_size();

    }
    else if(layer_type == "flatten")
    {
      this->get_output_shape(shape,i-1);

      batch_size = shape[0];
      rows = shape[1];
      columns = shape[2];
      channels = shape[3];
      //std::cout << "Setting up flatten layer - "<< batch_size <<" " << rows << " "<<columns <<" "<<channels << std::endl;

      Flatten * new_flat = new Flatten(batch_size,rows,columns,channels);
      layer_objects.push_back(new_flat);

      // layer_buffers[i]["input"] = layer_buffers[i-1]["output"];
      // layer_buffers[i]["dinput"] = layer_buffers[i-1]["doutput"];
      // layer_buffers[i]["output"] = layer_buffers[i]["input"];
      // layer_buffers[i]["doutput"] = layer_buffers[i]["dinput"];

      layer_buffer_bytes[i]["output"]=0;
      layer_buffer_bytes[i]["doutput"]=0;



    }
    else if(layer_type == "fc")
    {
      this->get_output_shape(shape,i-1);

      batch_size = shape[0];
      input_height = shape[1];
      output_height = atoi(layer_info[i][1].c_str());

      std::cout << "Setting up fc layer - "<< batch_size <<" " << input_height << std::endl;

      FCLayer * new_fc = new FCLayer(blas_handle,batch_size,input_height,output_height);

      bytes =  new_fc->get_output_shape_and_bytes(shape);



      layer_buffer_bytes[i]["output"]=bytes;//hipMalloc(&(layer_buffers[i]["output"]),bytes);
      layer_buffer_bytes[i]["doutput"]=bytes;//hipMalloc(&(layer_buffers[i]["output"]),bytes);


      // layer_buffers[i]["input"] = layer_buffers[i-1]["output"];
      // layer_buffers[i]["dinput"] = layer_buffers[i-1]["doutput"];


      //new_fc -> allocate_internal_mem(&(layer_buffers[i]["params"]),&(layer_buffers[i]["dparams"]));
      layer_buffer_bytes[i]["params"] = new_fc -> get_params_shape_and_bytes(shape);
      layer_buffer_bytes[i]["dparams"] = new_fc -> get_params_shape_and_bytes(shape);

      layer_objects.push_back(new_fc);

    }
    else if(layer_type == "softmax")
    {
      this->get_output_shape(shape,i-1);

      batch_size = shape[0];
      input_height = shape[1];

      Softmax * new_softmax = new Softmax(handle,batch_size,input_height);

      bytes =  new_softmax->get_output_shape_and_bytes(shape);

      layer_objects.push_back(new_softmax);

      //layer_buffers[i] = init_buffer_map();

      layer_buffer_bytes[i]["output"]=bytes;//hipMalloc(&(layer_buffers[i]["output"]),bytes);
      layer_buffer_bytes[i]["doutput"]=bytes;//hipMalloc(&(layer_buffers[i]["output"]),bytes);


      // layer_buffers[i]["input"] = layer_buffers[i-1]["output"];
      // layer_buffers[i]["dinput"] = layer_buffers[i-1]["doutput"];


    }
    else if(layer_type == "relu")
    {
      this->get_output_shape(shape,i-1);
      batch_size = shape[0];
      channels = shape[1];
      rows = shape[2];
      columns = shape[3];

      relu * new_relu =  new relu(handle,batch_size,channels,rows,columns);
      bytes = new_relu->get_output_shape_and_bytes(shape);

      layer_objects.push_back(new_relu);


      layer_buffer_bytes[i]["output"]=bytes;//hipMalloc(&(layer_buffers[i]["output"]),bytes);
      layer_buffer_bytes[i]["doutput"]=bytes;//hipMalloc(&(layer_buffers[i]["output"]),bytes);

      // layer_buffers[i]["input"] = layer_buffers[i-1]["output"];
      // layer_buffers[i]["dinput"] = layer_buffers[i-1]["doutput"];


    }
    else if(layer_type == "maxpool" || layer_type == "avgpool") {
      this->get_output_shape(shape, i-1);

      window_height = atoi(layer_info[i][1].c_str());
      window_width = atoi(layer_info[i][2].c_str());
      vertical_stride = atoi(layer_info[i][3].c_str());
      horizontal_stride = atoi(layer_info[i][4].c_str());
      pad = VALID;

      if (layer_type == "maxpool")
        pooling_type = HIPDNN_POOLING_MAX;
      else if (layer_type == "avgpool"){
        if (pad == VALID)
          pooling_type = HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
        else
          pooling_type = HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;
      }

      batch_size = shape[0];
      rows = shape[1];
      columns = shape[2];
      channels = shape[3];

      std::cout << "Setting up pooling layer - "<< batch_size <<" " << rows << " "<<columns <<" "<<channels << std::endl;

      PoolingLayer* new_pooling = new PoolingLayer(&handle,
        window_height,
        window_width,
        vertical_stride,
        horizontal_stride,
        batch_size,
        rows,
        columns,
        channels,
        pad,
        pooling_type
      );

      bytes =  new_pooling->get_output_shape_and_bytes(shape);
      layer_buffers[i] = init_buffer_map();
      //layer_buffers[i]["input"] = layer_buffers[i-1]["output"];
      layer_buffer_bytes[i]["output"]=bytes;//hipMalloc(&(layer_buffers[i]["output"]),bytes);

      //layer_buffers[i]["dinput"] = layer_buffers[i-1]["doutput"];
      layer_buffer_bytes[i]["doutput"]=bytes;//hipMalloc(&(layer_buffers[i]["output"]),bytes);


      layer_objects.push_back(new_pooling);
    }

  }
}


void seqNetwork::link_all_buffers()
{
  for(int i=1;i<num_layers;i++)
  {
    layer_buffers[i]["input"] = layer_buffers[i-1]["output"];
    layer_buffers[i]["dinput"] = layer_buffers[i-1]["doutput"];

    if(layer_info[i][0]=="flatten")
    {
        layer_buffers[i]["output"] = layer_buffers[i]["input"];
        layer_buffers[i]["doutput"] = layer_buffers[i]["dinput"];
    }

  }
}

void seqNetwork::randomise_batch()
{
  ((InputLayer*)layer_objects[0])->randomly_populate(layer_buffers[0]["output"],layer_buffers[0]["labels"]);
}

void seqNetwork::update_batch(float* data, int* labels)
{
  ((InputLayer*)layer_objects[0])->update_batch(data, (float*)labels,layer_buffers[0]["output"],layer_buffers[0]["labels"]);
}

void seqNetwork::randomise_params()
{
  for(int i=0;i<num_layers;i++)
  {
    if(layer_info[i][0]=="conv")
    {
      ((ConvLayer*)layer_objects[i])->populate_filter_params(layer_buffers[i]["params"]);
    }
    else if(layer_info[i][0]=="fc")
    {
      ((FCLayer*)layer_objects[i])->populate_filter_params(layer_buffers[i]["params"]);
    }
  }
}

void seqNetwork::forward()
{
  for(int i=0;i<num_layers;i++)
  {
    std::map<std::string,float*> buffer_map = layer_buffers[i];
    std::string layer_type = layer_info[i][0];

    //hipDeviceSynchronize();
    if(layer_type=="input")continue;
    else if(layer_type=="conv")
    {
      ConvLayer * layer_obj = (ConvLayer*)(layer_objects[i]);
      layer_obj -> forward(1.0,0.0,buffer_map["input"],buffer_map["params"],(void*)buffer_map["workspace"],buffer_map["output"]);
    }
    else if(layer_type=="fc")
    {
      FCLayer * layer_obj = (FCLayer*)(layer_objects[i]);
      layer_obj -> forward(buffer_map["input"],buffer_map["params"],buffer_map["output"]);
    }
    else if(layer_type == "softmax")
    {
      Softmax* layer_obj = (Softmax*)(layer_objects[i]);
      layer_obj -> forward(buffer_map["input"],buffer_map["output"]);
    }
    else if(layer_type == "maxpool" || layer_type == "avgpool"){
      PoolingLayer* layer_obj = (PoolingLayer*) (layer_objects[i]);
      layer_obj->forward(1.0,0.0,buffer_map["input"], buffer_map["output"]);
    }
    else if(layer_type=="relu")
    {
      relu * layer_obj = (relu*)(layer_objects[i]);
      layer_obj -> forward(buffer_map["input"],buffer_map["output"]);
    }

  }


}

 void seqNetwork::backward()
{

  for(int i=num_layers-1;i>=0;i--)
  {
    std::map<std::string,float*> buffer_map = layer_buffers[i];
    std::string layer_type = layer_info[i][0];
    //hipDeviceSynchronize();
    if(layer_type=="input")continue;
    else if(layer_type=="conv")
    {
      ConvLayer * layer_obj = (ConvLayer*)(layer_objects[i]);
      layer_obj -> backward(1.0,0.0,buffer_map["output"],buffer_map["doutput"],(void*)buffer_map["workspace"], buffer_map["params"], buffer_map["input"], buffer_map["dinput"], buffer_map["dparams"],lr);
    }
    else if(layer_type=="fc")
    {
      FCLayer * layer_obj = (FCLayer*)(layer_objects[i]);
      layer_obj -> backward(1.0,0.0,0.0,buffer_map["input"], buffer_map["params"],buffer_map["dparams"],buffer_map["dinput"], buffer_map["doutput"],lr);
    }
    else if(layer_type == "softmax")
    {
      Softmax* layer_obj = (Softmax*)(layer_objects[i]);
      layer_obj -> backward((int*)layer_buffers[0]["labels"],buffer_map["dinput"],buffer_map["output"]);
      //gradients are stored in buffer_map["labels"]
    }
    else if(layer_type == "maxpool" || layer_type == "avgpool")
    {
      PoolingLayer* layer_obj = (PoolingLayer*) (layer_objects[i]);
      layer_obj->backward(1.0,0.0,buffer_map["output"], buffer_map["doutput"] ,buffer_map["input"], buffer_map["dinput"]);
    }
    else if(layer_type=="relu")
    {
      relu * layer_obj = (relu*)(layer_objects[i]);
      layer_obj -> backward(buffer_map["input"],buffer_map["output"],buffer_map["dinput"],buffer_map["doutput"]);
    }
  }
}

void seqNetwork::update_weights() {
  for(int i=num_layers-1;i>=0;i--)
  {
    std::map<std::string,float*> buffer_map = layer_buffers[i];
    std::string layer_type = layer_info[i][0];
    //hipDeviceSynchronize();
    if(layer_type=="conv")
    {
      ConvLayer * layer_obj = (ConvLayer*)(layer_objects[i]);
      layer_obj -> update_weights( buffer_map["params"], buffer_map["dparams"],lr);
    }
    else if(layer_type=="fc")
    {
      FCLayer * layer_obj = (FCLayer*)(layer_objects[i]);
      layer_obj -> update_weights( buffer_map["params"],buffer_map["dparams"],lr);
    }

  }
}

float* seqNetwork::offload_buffer(int layer_number, std::string type,int shape[])
{
  int bytes;
  std::string layer_type = layer_info[layer_number][0];
  //std::cout << "Offloading " << layer_type << std::endl;
  if(layer_type=="conv")
  {
    ConvLayer * layer_obj = (ConvLayer*)(layer_objects[layer_number]);
    if(type=="output" || type=="doutput")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
    else if(type == "workspace")
      bytes = layer_obj->get_total_workspace_size();
    else if(type == "input" || type == "dinput")
      bytes = layer_obj->get_input_shape_and_bytes(shape);

  }
  else if(layer_type=="fc")
  {
    FCLayer * layer_obj = (FCLayer*)(layer_objects[layer_number]);
    if(type=="output" || type=="doutput")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
    else if(type == "input" || type == "dinput")
      bytes = layer_obj->get_input_shape_and_bytes(shape);
    else if(type=="params" || type == "dparams")
      bytes = layer_obj -> get_params_shape_and_bytes(shape);

  }
  else if(layer_type=="flatten")
  {

    Flatten * layer_obj = (Flatten*)(layer_objects[layer_number]);
    if(type=="output" || type=="doutput")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
    else if(type == "input" || type == "dinput")
      bytes = layer_obj->get_input_shape_and_bytes(shape);

  }
  else if(layer_type == "softmax")
  {

    Softmax * layer_obj = (Softmax*)(layer_objects[layer_number]);
    if(type=="output" || type=="doutput")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
    else if(type == "input" || type == "dinput")
      bytes = layer_obj->get_input_shape_and_bytes(shape);
  }
  else if(layer_type == "input")
  {
    InputLayer * layer_obj = (InputLayer*)(layer_objects[layer_number]);
    if(type=="output")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
  }
  else if (layer_type == "maxpool" || layer_type == "avgpool"){
    PoolingLayer * layer_obj = (PoolingLayer*)(layer_objects[layer_number]);
    if(type=="output")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
  }
  else if(layer_type == "relu")
  {
    relu * layer_obj = (relu*)(layer_objects[layer_number]);
    if(type == "output" || type == "doutput")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
    else if(type=="input" || type == "dinput")
      bytes = layer_obj->get_input_shape_and_bytes(shape);
  }

  if(layer_offloaded_buffers[layer_number][type] == nullptr){
    std::cout << "Allocating bytes to the layer buffer " << layer_number <<" " << type<<std::endl;
    layer_offloaded_buffers[layer_number][type] = (float*)malloc(bytes);

  }
  gpuErrchk(hipMemcpy(layer_offloaded_buffers[layer_number][type],layer_buffers[layer_number][type],bytes,
    hipMemcpyDeviceToHost));

  return layer_offloaded_buffers[layer_number][type];

  // hipFree(layer_buffers[layer_number][type]);
  // layer_buffers[layer_number][type] = nullptr;


}

void seqNetwork::prefetch_buffer(int layer_number,std::string type)
{
  int bytes,shape[4];
  std::string layer_type = layer_info[layer_number][0];
  std::cout << "Prefetching " << layer_type << std::endl;
  if(layer_type=="conv")
  {
    ConvLayer * layer_obj = (ConvLayer*)(layer_objects[layer_number]);
    if(type=="output")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
    else if(type == "workspace")
      bytes = layer_obj->get_total_workspace_size();
  }
  else if(layer_type=="fc")
  {
    FCLayer * layer_obj = (FCLayer*)(layer_objects[layer_number]);
    if(type=="output")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
  }
  else if(layer_type=="flatten")
  {
    std::cout << "Offloading " << layer_type << std::endl;
    Flatten * layer_obj = (Flatten*)(layer_objects[layer_number]);
    if(type=="output")
      bytes = layer_obj->get_output_shape_and_bytes(shape);

  }
  else if(layer_type == "softmax")
  {

    Softmax * layer_obj = (Softmax*)(layer_objects[layer_number]);
    if(type=="output")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
  }
  else if(layer_type == "input")
  {
    InputLayer * layer_obj = (InputLayer*)(layer_objects[layer_number]);
    if(type=="output")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
  }

  if(layer_buffers[layer_number][type] == nullptr)
    gpuErrchk(hipMalloc(&layer_buffers[layer_number][type],bytes));

  gpuErrchk(hipMemcpy(layer_buffers[layer_number][type],layer_offloaded_buffers[layer_number][type],bytes,
    hipMemcpyHostToDevice));



  //free(layer_offloaded_buffers[layer_number][type]);
  //layer_offloaded_buffers[layer_number][type] = nullptr;
}

void seqNetwork::allocate_all_memory(vmm * mem_manager)
{
  std::map<std::string,int>::iterator it;
  std::string buff_type;
  int buff_bytes;
  int total_bytes=0;
  for(int i=0;i<num_layers;i++)
  {
    std::cout << layer_info[i][0] << std::endl;
    it = layer_buffer_bytes[i].begin();
    while(it!=layer_buffer_bytes[i].end())
    {
      buff_type = it->first;
      buff_bytes = it->second;
      total_bytes += buff_bytes;
      //std::cout << buff_type << " " << buff_bytes << std::endl;
      mem_manager->allocate(&layer_buffers[i][buff_type],buff_bytes);
      it++;
    }
  }
  std::cout << total_bytes << std::endl;
  link_all_buffers();
}

void seqNetwork::allocate_mem_params(vmm * mem_manager)
{
  int bytes;
  for(int i=0;i<num_layers;i++)
  {
    if(layer_info[i][0] == "conv" || layer_info[i][0] == "fc")
    {
      bytes = layer_buffer_bytes[i]["params"];
      mem_manager->allocate(&layer_buffers[i]["params"],bytes,layer_info[i][0]+" params");

      bytes = layer_buffer_bytes[i]["dparams"];
      mem_manager->allocate(&layer_buffers[i]["dparams"],bytes,layer_info[i][0]+" dparams");

    }
  }
}

void seqNetwork::allocate_mem_layer(int layer_number, vmm * mem_manager)
{
  int i = layer_number,bytes;

  if(layer_info[i][0]!="flatten"){
    assert(layer_buffers[i]["output"] == nullptr);
    bytes = layer_buffer_bytes[i]["output"];
    mem_manager->allocate(&layer_buffers[i]["output"],bytes,layer_info[i][0]+" layer - output");
  }

  if(layer_info[i][0] == "input")
  {
    //allocate labels memory
    assert(layer_buffers[i]["labels"] == nullptr);
    bytes = layer_buffer_bytes[i]["labels"];
    mem_manager->allocate(&layer_buffers[i]["labels"],bytes,"input layer - labels");
  }

}

void seqNetwork::link_layer_buffer(int layer_number)
{
  int i = layer_number;
  if(i < num_layers-1)
  {
    layer_buffers[i+1]["input"] = layer_buffers[i]["output"];
  }
}


seqNetwork::~seqNetwork()
{
  hipdnnDestroy(handle);
  hipblasDestroy(blas_handle);
  for(int i=0;i<num_layers;i++)
  {
    if(layer_buffers[i]["input"]!=nullptr)
      hipFree(layer_buffers[i]["input"]);
    if(layer_buffers[i]["workspace"]!=nullptr)
      hipFree(layer_buffers[i]["workspace"]);
    if(layer_buffers[i]["output"]!=nullptr)
      hipFree(layer_buffers[i]["output"]);
    if(layer_buffers[i]["params"]!=nullptr)
      hipFree(layer_buffers[i]["params"]);

    if(layer_info[i][0]=="input")
      hipFree(layer_buffers[i]["labels"]);

  }
}
