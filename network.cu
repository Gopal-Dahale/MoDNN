#include "layers.h"
#include "pooling_layer.h"

using namespace network;
using namespace layers;

seqNetwork::seqNetwork(hipdnnHandle_t cudnn,hipblasHandle_t cublas,std::vector<std::string> &specs)
{
  /*
  Specs is a vector of strings specifying the Neural Network.
  Input -> "input N H W C"
  Conv ->  "conv H W C"
  */
  num_layers = specs.size();
  handle = cudnn;
  blas_handle = cublas;
  for(int i=0;i<num_layers;i++)
    {
      std::stringstream ss(specs[i]);
      std::vector<std::string> info;
      std::string tok;
      while(getline(ss, tok, ' ')){
        info.push_back(tok);
      }
      layer_info.push_back(info);
    }
}

void seqNetwork::print_network_info()
{
  for(int i=0;i<num_layers;i++)
  {
    std::cout << "Layer "<<i+1<<" : "<<layer_info[i][0] << std::endl;
    std::cout << "Specs : ";
    for(int j=1;j<layer_info[i].size();j++)
      std::cout << layer_info[i][j] << " ";
    std::cout << std::endl;
  }
}

void seqNetwork::get_output_shape(int shape[],int i)
{
  Layer *last_layer = layer_objects[i];
  if (layer_info[i][0] == "flatten")
    ((Flatten*)last_layer)->get_output_shape_and_bytes(shape);
  else if(layer_info[i][0] == "conv")
    ((ConvLayer*)last_layer) -> get_output_shape_and_bytes(shape);
  else if(layer_info[i][0] == "fc")
    ((FCLayer*)last_layer) -> get_output_shape_and_bytes(shape);
  else if(layer_info[i][0] == "softmax")
    ((Softmax*)last_layer) -> get_output_shape_and_bytes(shape);
  else if(layer_info[i][0] == "input")
    ((InputLayer*)last_layer) -> get_output_shape_and_bytes(shape);
  else if(layer_info[i][0] == "maxpool" || layer_info[i][0] == "avgpool" )
    ((PoolingLayer*)last_layer)->get_output_shape_and_bytes(shape);
}

void seqNetwork::allocate_memory()
{
  std::string layer_type;
  int shape[4],batch_size,rows,columns,channels;
  int kernel_rows,kernel_cols,kernel_channels,bytes;
  int input_height,output_height;
  int window_height, window_width;
  int vertical_stride, horizontal_stride;
  int pad, pooling_type;

  std::cout << "Allocating memory for the Neural Network" << std::endl;
  layer_buffers.resize(num_layers);
  layer_offloaded_buffers.resize(num_layers);

  for(int i=0;i<num_layers;i++)
  {
    layer_type = layer_info[i][0];
    std::cout << "Layer "<<i+1<<" : "<<layer_type << std::endl;
    layer_offloaded_buffers[i] = init_buffer_map();
    if(layer_type == "input")
    {
      batch_size = atoi(layer_info[i][1].c_str());
      rows = atoi(layer_info[i][2].c_str());
      columns = atoi(layer_info[i][3].c_str());
      channels = atoi(layer_info[i][4].c_str());

      InputLayer * new_ip = new InputLayer(batch_size,rows,columns,channels);
      layer_objects.push_back(new_ip);

      bytes = new_ip->get_output_shape_and_bytes(shape);
      layer_buffers[i] = init_buffer_map();
      hipMalloc(&(layer_buffers[i]["output"]),bytes);


    }
    else if(layer_type == "conv")
    {
      //batch_size is already fixed in the first input layer
      //std::cout << "Allocating Memory to Conv Layer" << std::endl;
      kernel_rows = atoi(layer_info[i][1].c_str());
      kernel_cols = atoi(layer_info[i][2].c_str());
      kernel_channels = atoi(layer_info[i][3].c_str());

      this->get_output_shape(shape,i-1);

      batch_size = shape[0];
      rows = shape[1];
      columns = shape[2];
      channels = shape[3];

      ConvLayer * new_conv = new ConvLayer(handle,batch_size,rows,columns,channels,kernel_rows,kernel_cols,kernel_channels,VALID);

      bytes =  new_conv->get_output_shape_and_bytes(shape);

      layer_objects.push_back(new_conv);

      layer_buffers[i] = init_buffer_map();
      hipMalloc(&(layer_buffers[i]["output"]),bytes);
      layer_buffers[i]["input"] = layer_buffers[i-1]["output"];
      new_conv -> allocate_internal_mem(&(layer_buffers[i]["params"]),(void**)&(layer_buffers[i]["workspace"]));

    }
    else if(layer_type == "flatten")
    {
      this->get_output_shape(shape,i-1);

      batch_size = shape[0];
      rows = shape[1];
      columns = shape[2];
      channels = shape[3];
      std::cout << "Setting up flatten layer - "<< batch_size <<" " << rows << " "<<columns <<" "<<channels << std::endl;

      Flatten * new_flat = new Flatten(batch_size,rows,columns,channels);
      layer_objects.push_back(new_flat);

      layer_buffers[i]["input"] = layer_buffers[i-1]["output"];
      layer_buffers[i]["output"] = layer_buffers[i]["input"];
    }
    else if(layer_type == "fc")
    {
      this->get_output_shape(shape,i-1);

      batch_size = shape[0];
      input_height = shape[1];
      output_height = atoi(layer_info[i][1].c_str());

      FCLayer * new_fc = new FCLayer(blas_handle,batch_size,input_height,output_height);

      bytes =  new_fc->get_output_shape_and_bytes(shape);



      layer_buffers[i] = init_buffer_map();
      hipMalloc(&(layer_buffers[i]["output"]),bytes);
      layer_buffers[i]["input"] = layer_buffers[i-1]["output"];
      new_fc -> allocate_internal_mem(&(layer_buffers[i]["params"]));


      layer_objects.push_back(new_fc);

    }
    else if(layer_type == "softmax")
    {
      this->get_output_shape(shape,i-1);

      batch_size = shape[0];
      input_height = shape[1];

      Softmax * new_softmax = new Softmax(handle,batch_size,input_height);

      bytes =  new_softmax->get_output_shape_and_bytes(shape);

      layer_objects.push_back(new_softmax);

      layer_buffers[i] = init_buffer_map();
      hipMalloc(&(layer_buffers[i]["output"]),bytes);
      layer_buffers[i]["input"] = layer_buffers[i-1]["output"];


    }
    else if(layer_type == "maxpool" || layer_type == "avgpool") {
      this->get_output_shape(shape, i-1);
      
      window_height = atoi(layer_info[i][1].c_str());
      window_width = atoi(layer_info[i][2].c_str());
      vertical_stride = atoi(layer_info[i][3].c_str());
      horizontal_stride = atoi(layer_info[i][4].c_str());
      pad = VALID;

      if (layer_type == "maxpool")
        pooling_type = HIPDNN_POOLING_MAX;
      else if (layer_type == "avgpool"){
        if (pad == VALID)
          pooling_type = HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
        else
          pooling_type = HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;
      }

      batch_size = shape[0];
      rows = shape[1];
      columns = shape[2];
      channels = shape[3];

      PoolingLayer* new_pooling = new PoolingLayer(handle, 
        window_height, 
        window_width,
        vertical_stride,
        horizontal_stride,
        batch_size,
        rows,
        columns,
        channels,
        pad,
        pooling_type
      );

      bytes =  new_pooling->get_output_shape_and_bytes(shape);
      layer_buffers[i] = init_buffer_map();
      layer_buffers[i]["input"] = layer_buffers[i-1]["output"];
      hipMalloc(&(layer_buffers[i]["output"]),bytes);

      layer_objects.push_back(new_pooling);
    }

  }
}

void seqNetwork::randomise_input()
{
  ((InputLayer*)layer_objects[0])->randomly_populate(layer_buffers[0]["output"]);
}

void seqNetwork::randomise_params()
{
  for(int i=0;i<num_layers;i++)
  {
    if(layer_info[i][0]=="conv")
    {
      ((ConvLayer*)layer_objects[i])->populate_filter_params(layer_buffers[i]["params"]);
    }
    else if(layer_info[i][0]=="fc")
    {
      ((FCLayer*)layer_objects[i])->populate_filter_params(layer_buffers[i]["params"]);
    }
  }
}

void seqNetwork::forward()
{
  for(int i=0;i<num_layers;i++)
  {
    std::map<std::string,float*> buffer_map = layer_buffers[i];
    std::string layer_type = layer_info[i][0];

    if(layer_type=="input")continue;
    else if(layer_type=="conv")
    {
      ConvLayer * layer_obj = (ConvLayer*)(layer_objects[i]);
      layer_obj -> forward(1.0,0.0,buffer_map["input"],buffer_map["params"],(void*)buffer_map["workspace"],buffer_map["output"]);
    }
    else if(layer_type=="fc")
    {
      FCLayer * layer_obj = (FCLayer*)(layer_objects[i]);
      layer_obj -> forward(buffer_map["input"],buffer_map["params"],buffer_map["output"]);
    }
    else if(layer_type == "softmax")
    {
      Softmax* layer_obj = (Softmax*)(layer_objects[i]);
      layer_obj -> forward(buffer_map["input"],buffer_map["output"]);
    }
    else if(layer_type == "maxpool" || layer_type == "avgpool"){
      PoolingLayer* layer_obj = (PoolingLayer*) (layer_objects[i]);
      layer_obj->forward(1.0,0.0,buffer_map["input"], buffer_map["output"]);
    }
  }
}

void seqNetwork::offload_buffer(int layer_number, std::string type)
{
  int bytes,shape[4];
  std::string layer_type = layer_info[layer_number][0];
  std::cout << "Offloading " << layer_type << std::endl;
  if(layer_type=="conv")
  {
    ConvLayer * layer_obj = (ConvLayer*)(layer_objects[layer_number]);
    if(type=="output")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
    else if(type == "workspace")
      bytes = layer_obj->get_total_workspace_size();
  }
  else if(layer_type=="fc")
  {
    FCLayer * layer_obj = (FCLayer*)(layer_objects[layer_number]);
    if(type=="output")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
  }
  else if(layer_type=="flatten")
  {
    std::cout << "Offloading " << layer_type << std::endl;
    Flatten * layer_obj = (Flatten*)(layer_objects[layer_number]);
    if(type=="output")
      bytes = layer_obj->get_output_shape_and_bytes(shape);

  }
  else if(layer_type == "softmax")
  {

    Softmax * layer_obj = (Softmax*)(layer_objects[layer_number]);
    if(type=="output")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
  }
  else if(layer_type == "input")
  {
    InputLayer * layer_obj = (InputLayer*)(layer_objects[layer_number]);
    if(type=="output")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
  }
  else if (layer_type == "maxpool" || layer_type == "avgpool"){
    PoolingLayer * layer_obj = (PoolingLayer*)(layer_objects[layer_number]);
    if(type=="output")
      bytes = layer_obj->get_output_shape_and_bytes(shape);
  }

  if(layer_offloaded_buffers[layer_number][type] == nullptr)
    layer_offloaded_buffers[layer_number][type] = (float*)malloc(bytes);

  hipMemcpy(layer_offloaded_buffers[layer_number][type],layer_buffers[layer_number][type],bytes,
    hipMemcpyDeviceToHost);


}

void seqNetwork::prefetch_buffer(int layer_number,std::string type)
{

}
